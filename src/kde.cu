/**
 * kde.cu: kde calculation methods, including the CUDA kernel function
 * Authors: Yizhao Gao <ygao29@illinois.edu>
 * Date: {08/19/2015}
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE 16

__global__ void kdeKernel(float * dDensity, float * dX, float * dY, int * dIndex, int nXCell, int nYCell, float xMin, float yMax, float cellSize, float bandwidth2, int blockBandwidth)
{
	__shared__ float sX[BLOCKSIZE * BLOCKSIZE];
	__shared__ float sY[BLOCKSIZE * BLOCKSIZE];

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	int idInThread = threadIdx.y * blockDim.x + threadIdx.x;

	float cellX = xMin + cellSize * (j + 0.5); 
	float cellY = yMax - cellSize * (i + 0.5);

	float density = 0.0f;
	float dist2;

	int pointProcessed;
	int pointToProcess;
	int endPoint;

	for(int k = 0; k < 1 + 2 * blockBandwidth; k ++)
	{
		int dataBID = (blockIdx.y + k) * (gridDim.x + 2 * blockBandwidth) + blockIdx.x;
		if(dataBID < 1)
		{
			pointProcessed = 0;
		}	
		else
		{
			pointProcessed = dIndex[dataBID - 1];
		}
		endPoint = dIndex[dataBID + 2 * blockBandwidth];

		pointToProcess = BLOCKSIZE * BLOCKSIZE;

		for(; pointProcessed < endPoint; pointProcessed += BLOCKSIZE * BLOCKSIZE)
		{
			if(pointProcessed + pointToProcess > endPoint)
			{	
				pointToProcess = endPoint - pointProcessed;
			}

			if(idInThread < pointToProcess)
			{
				sX[idInThread] = dX[pointProcessed + idInThread];
				sY[idInThread] = dY[pointProcessed + idInThread];
			}
			__syncthreads();

			for(int m = 0; m < pointToProcess; m++)
			{
				dist2 = (cellX - sX[m]) * (cellX - sX[m]) + (cellY - sY[m]) * (cellY - sY[m]);
				if(dist2 < bandwidth2)
				{
					density += (1 - dist2/bandwidth2);
				}
			}
			
			__syncthreads();
		}
	}

	
	if(i < nYCell && j < nXCell && i > -1 && j > -1)
	{
		dDensity[i * nXCell + j] = density * 2 / (M_PI * bandwidth2) * cellSize * cellSize;
		//dDensity[i * nXCell + j] = density;
	}
}

float * performKDE(float * xCor, float * yCor, int nPoints, int * pointIndex, int gridX, int gridY, int blockBandwidth, int cellX, int cellY, float xMin, float yMax, float cellSize, float bandwidth)
{
	hipError_t err;

	dim3 dimBlock (BLOCKSIZE, BLOCKSIZE);
	dim3 dimGrid (gridX, gridY);

	float * myDensity;
	
	if(NULL == (myDensity = (float *) malloc(sizeof(float) * cellX * cellY)))
	{
		printf("ERROR: Out of memory at %d in file %s!\n", __LINE__, __FILE__);
		exit(1);
	}

	int dataGridX = gridX + 2 * blockBandwidth;
	int dataGridY = gridY + 2 * blockBandwidth;

	float * dX;
	float * dY;
	int * dIndex;
	float * dDensity;


	err = hipMalloc((void **)&dX, sizeof(float) * nPoints);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	err = hipMalloc((void **)&dY, sizeof(float) * nPoints);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}

	err = hipMalloc((void **)&dIndex, sizeof(int) * dataGridX * dataGridY);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	err = hipMalloc((void **)&dDensity, sizeof(float) * cellX * cellY);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}

	err = hipMemcpy(dX, xCor, sizeof(float) * nPoints, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	err = hipMemcpy(dY, yCor, sizeof(float) * nPoints, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	err = hipMemcpy(dIndex, pointIndex, sizeof(int) * dataGridX * dataGridY, hipMemcpyHostToDevice);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}


	//KDE kernel
	kdeKernel<<<dimGrid,dimBlock>>>(dDensity, dX, dY, dIndex, cellX, cellY, xMin, yMax, cellSize, bandwidth * bandwidth, blockBandwidth);

	err = hipMemcpy(myDensity, dDensity, sizeof(float) * cellX * cellY, hipMemcpyDeviceToHost);
	if(err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(1);
	}
	
	hipFree(dIndex);
	hipFree(dDensity);

	hipFree(dX);
	hipFree(dY);

	return myDensity;
}
